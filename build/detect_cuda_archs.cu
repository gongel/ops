#include "stdio.h"
#include "hip/hip_runtime.h"

int main() {
  int count = 0;
  if (hipSuccess != hipGetDeviceCount(&count)) return -1;
  if (count == 0) return -1;
  for (int device = 0; device < count; ++device) {
    hipDeviceProp_t prop;
    if (hipSuccess == hipGetDeviceProperties(&prop, device))
      printf("%d.%d ", prop.major, prop.minor);
  }
  return 0;
}
